
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 32

__global__ 
void thread_multi(int *t1)
{
    int i=blockDim.x * blockIdx.x + threadIdx.x;
    int j=threadIdx.x;
    t1[i]=j;
}

int main()
{
    int *d_t1;
    int *h_t1;
    int i=0;
    
    size_t size = N*sizeof(float);
    
    //Memory allocation
    hipMalloc((void **)&d_t1, size);
    h_t1=(int *)malloc(size);
    
    //Initialize the array
    for (i=0; i<N; i++)
        h_t1[i] = 0;

    //Run invoke threads to execute the kernel
    hipMemcpy(d_t1, h_t1, size, hipMemcpyHostToDevice);
    thread_multi<<<4,N/4>>>(d_t1);
    hipMemcpy(h_t1, d_t1, size, hipMemcpyDeviceToHost);

    for(i=0; i<N; i++)
        printf("%d: %d\n",i, h_t1[i]);

    hipFree(d_t1);
    free(h_t1);
    printf("\nDone\n");

    return 0;
}

