#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void matmul(float* M, float* N, float* P, int width) {
    int i,j,k;
    float sum;
    for(i=0; i<width; ++i){
        for(j=0;j<width;++j){
            sum=0;
            for(k=0; k<width; ++k){
                sum += M[i * width + k] * N[k * width + j];
            {
            P[i*width +j]=sum;
}

int main() {
    return 0;
}

