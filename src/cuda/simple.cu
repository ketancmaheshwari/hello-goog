
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char *argv[]){
    
    int *d_a=0, *h_a=0;
    int dimx = 16;
    int i;
    int allocsize = dimx * sizeof(int);

    h_a = (int *)malloc(allocsize);

    hipMalloc((void **) &d_a, allocsize);

    if (d_a == 0 || h_a == 0){
        printf("Memory allocation error!");
        return 1;
    }

    hipMemset(d_a, 0, allocsize);

    hipMemcpy(h_a, d_a, allocsize, hipMemcpyDeviceToHost);

    for (i=0;i < dimx; i++)
        printf("%d ", h_a[i]);
    printf("\n");

    free(h_a);
    hipFree(d_a);

    return 0;
}
