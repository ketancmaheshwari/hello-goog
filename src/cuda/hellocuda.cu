
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void hellocuda(int* tidx ){
   /* *tidx = 100; */
   int x = threadIdx.x;
   tidx[x]=threadIdx.x;
}

int main(){
    
    int i;
    int * d_tidx;

    int * h_tidx;
    
    hipError_t err = hipMalloc((void**) &d_tidx, 20*sizeof(int));
    if (err != hipSuccess){
        printf("%s on %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    h_tidx=(int *)malloc(20*sizeof(int));

    hellocuda<<<200,20,20>>>(d_tidx);
    
    hipMemcpy(h_tidx, d_tidx, 20*sizeof(int), hipMemcpyDeviceToHost);

    for (i=0; i<20;i++)
        printf("Thread index: %d\n", h_tidx[i]);
    
    /*
    printf("Number of Blocks (Grid dim): %d\n", *h_gdim);
    printf("Number of Threads in current block (Block dim): %d\n", *h_bdim);
    printf("Block index: %d\n", *h_bidx);
    printf("Thread index: %d\n", *h_tidx);
    */
    return EXIT_SUCCESS;
}
